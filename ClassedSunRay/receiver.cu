#include "hip/hip_runtime.h"
#include "receiver.cuh"

// atomicAdd can only occur in .cu file
// but __device__ function can only in the same compile unit

//__device__ void add_energy(float *d_image, 
//	const float &u, const float &v, 
//	const int2 &resolution,
//	const float &energy)
//{
//	int2 row_col = make_int2(u* resolution.y, v* resolution.x);	// Intersect location																	  
//	int address = row_col.x*resolution.x + row_col.y;				//col_row.y + col_row.x*resolution.y;																	
//	atomicAdd(&(d_image[address]), energy);
//}
//
//inline __device__ void Receiver::GAddEnergy(const float &u, const float &v, const float &energy)	// add energy to d_image																							
//{
//	int2 row_col = make_int2(u* resolution_.y, v* resolution_.x);	// Intersect location																	  
//	int address = row_col.x*resolution_.x + row_col.y;				//col_row.y + col_row.x*resolution.y;																	
//	atomicAdd(&(d_image_[address]), energy);
//}

//inline __device__ bool RectangleReceiver::GIntersect(const float3 &orig, const float3 &dir, float &t, float &u, float &v)
//{ 
//	return global_func::rayParallelogramIntersect(orig, dir, rect_vertex_[0], rect_vertex_[1], rect_vertex_[3], t, u, v);	
//}
