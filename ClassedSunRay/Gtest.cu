#include "hip/hip_runtime.h"
#include "Gtest.cuh"
 
void test(SolarScene &solar_scene)
{
	//Heliostat *h = new RectangleHelio[4];
	// Receiver
	//solar_scene.receivers[0]->normal_ = make_float3(1.0f, 0.0f, 0.0f);
	solar_scene.receivers[0]->CInit(100);
	
	float3 dir = make_float3(0.0f, 0.0f, - 1.0f);
	float3 orig = make_float3(4.0f, 104.0f, 100.0f);

	float t, u, v;
	solar_scene.receivers[0]->GIntersect(orig, dir, t, u, v);
	
	// Heliostat
	RectangleHelio *rectangle_helio =dynamic_cast<RectangleHelio *>(solar_scene.heliostats[1]);
	rectangle_helio->normal_ = make_float3(0, 0, 1);
	rectangle_helio->Cset_sub_row_col(0.01f);
	rectangle_helio->Cinit(solar_scene.receivers[0]->pos_);

	// Grid
	RectGrid *rectgrid = dynamic_cast<RectGrid *>(solar_scene.grid0s[0]);
	rectgrid->CGridHelioMatch(solar_scene.heliostats);
}