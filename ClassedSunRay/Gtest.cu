#include "hip/hip_runtime.h"
#include "Gtest.cuh"

__global__ void add(float u, float v)
{

}
 
void test(SolarScene &solar_scene)
{
	// test receiver.cuh
	//solar_scene.receivers[0]->normal_ = make_float3(1.0f, 0.0f, 0.0f);
	solar_scene.receivers[0]->CInit(100);
	
	float3 dir = make_float3(0.0f, 0.0f, - 1.0f);
	float3 orig = make_float3(4.0f, 104.0f, 100.0f);

	// CPU
	float t, u, v;
	solar_scene.receivers[0]->GIntersect(orig, dir, t, u, v);
	
	// 
	RectangleHelio *rectangle_helio =dynamic_cast<RectangleHelio *>(solar_scene.heliostats[0]);
	rectangle_helio->Cset_sub_row_col(0.01f);
	rectangle_helio->init(solar_scene.receivers[0]->pos_);
}