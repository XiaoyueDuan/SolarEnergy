#include "hip/hip_runtime.h"
#include "Gtest.cuh"
#include "scene_instance_process.h"
#include "recthelio_tracing.h"
#include "image_save.h"
#include "image_smooth.cuh"

#include <sstream>


namespace tmp
{
	template<typename T>
	void init_matrix(T *matrix, int size)
	{
		for (int i = 0; i < size; ++i)
			matrix[i] = 0;
	}

	void save_array(string filename, float *array, int size)
	{
		ofstream out(filename.c_str());
		for (int i = 0; i < size; ++i)
			out << array[i] << endl;
		out.close();
	}
};

void test(SolarScene &solar_scene)
{
	////// helios
	////float helio_granularity[] = { 0.01f, 0.02f };
	//
	////// rays
	////int sun_shape_per_group[] = { 1024, 2048 };
	////float csrs[] = { 0.1f, 0.15f, 0.2f };	
	////float disturb_stds[] = { 0.001,0.002,0.003 };
	//
	//// helios
	//float helio_granularity[] = { 0.01f };
	////float helio_granularity[] = { 0.02f, 0.05f, 0.1f };
	//
	//// rays
	//int sun_shape_per_group[] = { 128 };
	////int sun_shape_per_group[] = {512};
	//float csrs[] = { 0.1f };
	////float csrs[] = { 0.2f, 0.3f };
	//float disturb_stds[] = { 0.001 };
	////float disturb_stds[] = { 0.003 };
	//
	//int start_n = 0, end_n = 50, save_number = 2000;
	//string save_path("../result//try//128-128-_____.txt");
	//
	//float *h_image = nullptr;
	//
	//int index = 0;
	//Receiver *recv = dynamic_cast<RectangleReceiver *>(solar_scene.receivers[0]);
	//
	//
	//float *h_avg_img = new float[recv->resolution_.x*recv->resolution_.y];
	//tmp::init_matrix(h_avg_img, recv->resolution_.x*recv->resolution_.y);
	//
	////int helio_num[] = { 4, 14, 24, 34,
	////					20, 22, 26, 28, 29 };
	//int helio_num[] = { 24 };
	//
	//for (int id = 0; id < sizeof(helio_num) / sizeof(helio_num[0]); ++id)
	//{
	//	int i = helio_num[id];
	//	tmp::init_matrix(h_avg_img, recv->resolution_.x*recv->resolution_.y);
	//
	//	RectangleHelio *recthelio = dynamic_cast<RectangleHelio *>(solar_scene.heliostats[i]);
	//
	//	int avg_index = 0;
	//	for (int i_gral = 0; i_gral < sizeof(helio_granularity) / sizeof(float); ++i_gral)
	//	{
	//		recthelio->pixel_length_ = helio_granularity[i_gral];
	//		for (int j = start_n; j < end_n; ++j)
	//		{
	//			for (int i_n_per_group = 0; i_n_per_group < sizeof(sun_shape_per_group) / sizeof(int); ++i_n_per_group)
	//			{
	//				solar_scene.sunray_->num_sunshape_lights_per_group_ = sun_shape_per_group[i_n_per_group];
	//				solar_scene.sunray_->CClear();
	//				for (int i_csr = 0; i_csr < sizeof(csrs) / sizeof(float); ++i_csr)
	//				{
	//					solar_scene.sunray_->csr_ = csrs[i_csr];
	//					for (int i_dist = 0; i_dist < sizeof(disturb_stds) / sizeof(float); ++i_dist)
	//					{
	//						solarenergy::disturb_std = disturb_stds[i_dist];
	//						// reset sunray
	//						SceneProcessor::set_sunray_content(*solar_scene.sunray_);
	//
	//						// clean screen to all 0s
	//						recv->Cclean_image_content();
	//
	//						// ray-tracing
	//						recthelio_ray_tracing(*solar_scene.sunray_,
	//							*recv,
	//							*recthelio,
	//							*solar_scene.grid0s[i],
	//							solar_scene.heliostats);
	//
	//						// Save result
	//						global_func::gpu2cpu(h_image, recv->d_image_, recv->resolution_.x*recv->resolution_.y);
	//						// Id, Ssub, rou, Nc
	//						float Id = solar_scene.sunray_->dni_;
	//						float Ssub = recthelio->pixel_length_*recthelio->pixel_length_;
	//						float rou = solarenergy::reflected_rate;
	//						int Nc = solar_scene.sunray_->num_sunshape_lights_per_group_;
	//						float Srec = recv->pixel_length_*recv->pixel_length_;
	//						float max = -1.0f, max_avg = -1.0f;
	//						for (int p = 0; p < recv->resolution_.x*recv->resolution_.y; ++p)
	//						{
	//							h_image[p] = h_image[p] * Id * Ssub * rou / Nc / Srec;
	//							h_avg_img[p] = (h_avg_img[p] * float(j) + h_image[p]) / float(j + 1);
	//
	//							if (max < h_image[p])
	//								max = h_image[p];
	//
	//							if (max_avg < h_avg_img[p])
	//								max_avg = h_avg_img[p];
	//						}
	//
	//						if (j < save_number)
	//						{
	//							string tmp = save_path;
	//							tmp.insert(tmp.size() - 9, to_string(i));
	//							tmp.insert(tmp.size() - 8, to_string(j));
	//							tmp.insert(tmp.size() - 7, to_string(helio_granularity[i_gral]));
	//							tmp.insert(tmp.size() - 6, to_string(sun_shape_per_group[i_n_per_group]));
	//							tmp.insert(tmp.size() - 5, to_string(csrs[i_csr]));
	//							tmp.insert(tmp.size() - 4, to_string(disturb_stds[i_dist]));
	//							ImageSaver::savetxt(tmp, recv->resolution_.x, recv->resolution_.y, h_image);
	//						}
	//						
	//						printf("No.%d\n", ++index);
	//					}
	//				}
	//			}
	//			printf("(%d,\t%d)\n", i, j);
	//		}
	//	}
	//}
	//
	//delete[] h_image;
	//h_image = nullptr;
	//
	//delete[] h_avg_img;
	//h_avg_img = nullptr;

	string save_path("../result/24////24-.txt"); // e.g. - ../result/24/256/24-0.txt
	int helio_id[] = { 24 };
	int num_lights[] = { 256, 512,1024,2048 };
	SubCenterType heliocenters[] = { SubCenterType::Grid, SubCenterType::Poisson };
	string heliocenters_name[] = { "grid_center", "poisson_center" };
	int run_times_start = 0, run_times_end = 100;

	// Smooth result
	int kernel_radius = 5;
	float trimmed_ratio = 0.02;
	float *h_image = nullptr;

	RectangleHelio *recthelio = dynamic_cast<RectangleHelio *>(solar_scene.heliostats[24]);
	Receiver *recv = dynamic_cast<RectangleReceiver *>(solar_scene.receivers[0]);
	
	int N = 256 * 256;
	// 256-256 128-512 64-1024 32-2048
	for (int j = run_times_start; j < run_times_end; ++j)
	{
		solar_scene.sunray_->CClear();
		solar_scene.sunray_->num_sunshape_groups_ = 1;
		solar_scene.sunray_->num_sunshape_lights_per_group_ = N;
		SceneProcessor::set_sunray_content(*solar_scene.sunray_);
		for (int i = 0; i < sizeof(num_lights) / sizeof(num_lights[0]); ++i)
		{
			solar_scene.sunray_->num_sunshape_lights_per_group_ = num_lights[i];
			solar_scene.sunray_->num_sunshape_groups_ = N / num_lights[i];

			// Grid or Poisson
			for (int t = 0; t < sizeof(heliocenters) / sizeof(heliocenters[0]); ++t)
			{
				// Clear result of receiver
				recv->Cclean_image_content();
				recthelio->type = heliocenters[t];

				// Ray-tracing
				int num_subcenters = recthelio_ray_tracing(*solar_scene.sunray_,
					*recv, *recthelio,
					*solar_scene.grid0s[0],
					solar_scene.heliostats);
				global_func::gpu2cpu(h_image, solar_scene.receivers[0]->d_image_, recv->resolution_.x*recv->resolution_.y);

				// Non Smooth
				float Id = solar_scene.sunray_->dni_;
				float rou = solarenergy::reflected_rate;
				int Nc = solar_scene.sunray_->num_sunshape_lights_per_group_;

				int num_recv_m2 = (1 / solar_scene.receivers[0]->pixel_length_)*(1 / solar_scene.receivers[0]->pixel_length_);
				float w = recthelio->size_.x - recthelio->gap_.x*(recthelio->row_col_.y - 1);
				float h = recthelio->size_.z - recthelio->gap_.y*(recthelio->row_col_.x - 1);
				float multiplier = (w*h*float(num_recv_m2)*Id * rou) / float(Nc*num_subcenters);

				for (int k = 0; k < recv->resolution_.x*recv->resolution_.y; ++k)
					h_image[k] *= multiplier;

				// Save image	
				string tmp_path = save_path;//"../result/24////24-.txt"
				tmp_path.insert(tmp_path.size() - 10, heliocenters_name[t]);
				tmp_path.insert(tmp_path.size() - 9, to_string(N / num_lights[i])+"_"+ to_string(num_lights[i]));
				tmp_path.insert(tmp_path.size() - 8, "non_smooth");
				tmp_path.insert(tmp_path.size() - 4, to_string(j));
				ImageSaver::savetxt(tmp_path, recv->resolution_.x, recv->resolution_.y, h_image, 5);

				// Smooth
				ImageSmoother::image_smooth(recv->d_image_,
					kernel_radius, trimmed_ratio,
					recv->resolution_.x, recv->resolution_.y);
				global_func::gpu2cpu(h_image, solar_scene.receivers[0]->d_image_, recv->resolution_.x*recv->resolution_.y);
				for (int k = 0; k < recv->resolution_.x*recv->resolution_.y; ++k)
					h_image[k] *= multiplier;
				tmp_path = save_path;//"../result/24////24-.txt"
				tmp_path.insert(tmp_path.size() - 10, heliocenters_name[t]);
				tmp_path.insert(tmp_path.size() - 9, to_string(N / num_lights[i]) + "_" + to_string(num_lights[i]));
				tmp_path.insert(tmp_path.size() - 8, "smooth");
				tmp_path.insert(tmp_path.size() - 4, to_string(j));
				ImageSaver::savetxt(tmp_path, recv->resolution_.x, recv->resolution_.y, h_image, 5);
				cout << tmp_path << endl;
			}
		}
	}
	
	//RectangleHelio *recthelio = dynamic_cast<RectangleHelio *>(solar_scene.heliostats[24]);
	//int num_subcenters =recthelio_ray_tracing(*solar_scene.sunray_,
	//						*solar_scene.receivers[0],
	//						*recthelio,
	//						*solar_scene.grid0s[0],
	//						solar_scene.heliostats);
	//// Smooth result
	//int kernel_radius = 5;
	//float trimmed_ratio = 0.03;
	//ImageSmoother::image_smooth(solar_scene.receivers[0]->d_image_,
	//	kernel_radius, trimmed_ratio,
	//	solar_scene.receivers[0]->resolution_.x, solar_scene.receivers[0]->resolution_.y);

	//float *h_image = nullptr;
	//global_func::gpu2cpu(h_image, solar_scene.receivers[0]->d_image_, solar_scene.receivers[0]->resolution_.x*solar_scene.receivers[0]->resolution_.y);
	//// Id, Ssub, rou, Nc
	//float Id=solar_scene.sunray_->dni_;
	////float Ssub = recthelio->pixel_length_*recthelio->pixel_length_;
	//float rou = solarenergy::reflected_rate;
	//int Nc = solar_scene.sunray_->num_sunshape_lights_per_group_;
	////float Srec = solar_scene.receivers[0]->pixel_length_*solar_scene.receivers[0]->pixel_length_;
	//float max = -1.0f;

	//int num_recv_m2 = (1 / solar_scene.receivers[0]->pixel_length_)*(1 / solar_scene.receivers[0]->pixel_length_);
	//float w = recthelio->size_.x - recthelio->gap_.x*(recthelio->row_col_.y - 1);
	//float h = recthelio->size_.z - recthelio->gap_.y*(recthelio->row_col_.x - 1);
	//float multiplier = (w*h*float(num_recv_m2)*Id * rou) / float(Nc*num_subcenters);

	//for (int i = 0; i < solar_scene.receivers[0]->resolution_.x*solar_scene.receivers[0]->resolution_.y; ++i)
	//{
	//	//h_image[i] = h_image[i] * Id * Ssub * rou / Nc/ Srec;

	//	// Ssub / Srec = Shelio_area* (1/Srec) / num_subcenters
	//	h_image[i] *= multiplier;	
	//	if (max < h_image[i])
	//		max = h_image[i];
	//}
	//
	//// Save image	
	//ImageSaver::savetxt("../result/24th-64group-1024pergroup-poisson-smoothed.txt", solar_scene.receivers[0]->resolution_.x, solar_scene.receivers[0]->resolution_.y, h_image);
}