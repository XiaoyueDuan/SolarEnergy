#include "hip/hip_runtime.h"
#include "Gtest.cuh"
#include "scene_instance_process.h"
#include "recthelio_tracing.h"
 
void test(SolarScene &solar_scene)
{
	solar_scene.InitContent();
	
	RectangleHelio *recthelio = dynamic_cast<RectangleHelio *>(solar_scene.heliostats[4]);
	recthelio_ray_tracing(*solar_scene.sunray_,
							*solar_scene.receivers[0],
							*recthelio,
							*solar_scene.grid0s[1],
							solar_scene.heliostats);

	//// float3 *d_microhelio_centers
	//// float3 *d_microhelio_normals
	//// microhelio_num
	//size_t size;
	//float3 *d_microhelio_centers = nullptr;
	//float3 *d_microhelio_normals = nullptr;
	//
	//set_microhelio_centers(*recthelio, d_microhelio_centers, d_microhelio_normals, size);
	//hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

	//// float3 *d_helio_vertexs
	//int start_pos = solar_scene.grid0s[0]->start_helio_pos_;
	//int end_pos = start_pos + solar_scene.grid0s[0]->num_helios_;
	//float3 *d_helio_vertexs = nullptr;
	//set_helios_vertexes(solar_scene.heliostats, start_pos, end_pos, d_helio_vertexs);

	//// int *d_microhelio_groups
	//int *d_microhelio_groups = nullptr;
	//set_microhelio_groups(d_microhelio_groups, solar_scene.sunray_->num_sunshape_groups_, size);

	//int *h_microhelio_groups = nullptr;
	//global_func::gpu2cpu(h_microhelio_groups, d_microhelio_groups, size);
}