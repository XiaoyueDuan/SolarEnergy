#include "hip/hip_runtime.h"
#include "Gtest.cuh"
 
void test(SolarScene &solar_scene)
{
	//Heliostat *h = new RectangleHelio[4];
	// Receiver
	//solar_scene.receivers[0]->normal_ = make_float3(1.0f, 0.0f, 0.0f);
	//solar_scene.receivers[0]->CInit(100);
	//
	//float3 dir = make_float3(0.0f, 0.0f, - 1.0f);
	//float3 orig = make_float3(4.0f, 104.0f, 100.0f);

	//float t, u, v;
	//solar_scene.receivers[0]->GIntersect(orig, dir, t, u, v);
	//
	//// Heliostat
	//RectangleHelio *rectangle_helio =dynamic_cast<RectangleHelio *>(solar_scene.heliostats[1]);
	//rectangle_helio->normal_ = make_float3(0, 0, 1);
	//rectangle_helio->Cset_sub_row_col(0.01f);
	//rectangle_helio->Crotate(solar_scene.receivers[0]->pos_);

	//// Grid
	//RectGrid *rectgrid = dynamic_cast<RectGrid *>(solar_scene.grid0s[0]);
	//rectgrid->CGridHelioMatch(solar_scene.heliostats);

	// random generator
	///* cpu */
	int size = 100;
	float *h_array = new float[size];	
	int *h_array_int = new int[size];
	RandomGenerator::cpu_Uniform(h_array, size);
	RandomGenerator::cpu_Gaussian(h_array, 0.0f, 1.0f, size);	
	RandomGenerator::cpu_Uniform(h_array_int, -5, 5, size);

	/* gpu */
	float *d_array = nullptr;
	int *d_array_int = nullptr;
	checkCudaErrors(hipMalloc((void **)&d_array, size * sizeof(float)));
	checkCudaErrors(hipMalloc((void **)&d_array_int, size * sizeof(int)));
	RandomGenerator::gpu_Uniform(d_array, size);
	RandomGenerator::gpu_Gaussian(d_array, 0.0f, 1.0f, size);
	RandomGenerator::gpu_Uniform(d_array_int, -5, 5, size);
	float *h_array_aftercompute = nullptr;
	global_func::gpu2cpu(h_array_aftercompute, d_array, size);

	delete[] h_array;
	delete[] h_array_int;
	h_array = nullptr;
	h_array_int = nullptr;
	hipFree(d_array);
	d_array = nullptr;

}