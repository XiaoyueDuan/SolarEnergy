#include "hip/hip_runtime.h"
#include "Gtest.cuh"
#include "scene_instance_process.h"
 
void test(SolarScene &solar_scene)
{
	// 1
	SceneProcessor::set_grid_content(solar_scene.grid0s, solar_scene.heliostats);
	
	// 2
	float3 focus_center = solar_scene.receivers[0]->pos_;
	SceneProcessor::set_helio_content(solar_scene.heliostats, focus_center);

	// 3
	SceneProcessor::set_receiver_content(solar_scene.receivers);

	// 4
	SceneProcessor::set_sunray_content(*solar_scene.sunray_);
}