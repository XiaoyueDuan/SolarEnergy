#include "hip/hip_runtime.h"
#include "Gtest.cuh"
#include "scene_instance_process.h"
#include "recthelio_tracing.h"
#include "image_save.h"

#include <sstream>


namespace tmp
{
	template<typename T>
	void init_matrix(T *matrix, int size)
	{
		for (int i = 0; i < size; ++i)
			matrix[i] = 0;
	}

	void save_array(string filename, float *array, int size)
	{
		ofstream out(filename.c_str());
		for (int i = 0; i < size; ++i)
			out << array[i] << endl;
		out.close();
	}
};

void test(SolarScene &solar_scene)
{
	////// helios
	////float helio_granularity[] = { 0.01f, 0.02f };
	//
	////// rays
	////int sun_shape_per_group[] = { 1024, 2048 };
	////float csrs[] = { 0.1f, 0.15f, 0.2f };	
	////float disturb_stds[] = { 0.001,0.002,0.003 };
	//
	//// helios
	//float helio_granularity[] = { 0.01f };
	////float helio_granularity[] = { 0.02f, 0.05f, 0.1f };
	//
	//// rays
	//int sun_shape_per_group[] = { 128 };
	////int sun_shape_per_group[] = {512};
	//float csrs[] = { 0.1f };
	////float csrs[] = { 0.2f, 0.3f };
	//float disturb_stds[] = { 0.001 };
	////float disturb_stds[] = { 0.003 };
	//
	//int start_n = 0, end_n = 50, save_number = 2000;
	//string save_path("../result//try//128-128-_____.txt");
	//
	//float *h_image = nullptr;
	//
	//int index = 0;
	//Receiver *recv = dynamic_cast<RectangleReceiver *>(solar_scene.receivers[0]);
	//
	//
	//float *h_avg_img = new float[recv->resolution_.x*recv->resolution_.y];
	//tmp::init_matrix(h_avg_img, recv->resolution_.x*recv->resolution_.y);
	//
	////int helio_num[] = { 4, 14, 24, 34,
	////					20, 22, 26, 28, 29 };
	//int helio_num[] = { 24 };
	//
	//for (int id = 0; id < sizeof(helio_num) / sizeof(helio_num[0]); ++id)
	//{
	//	int i = helio_num[id];
	//	tmp::init_matrix(h_avg_img, recv->resolution_.x*recv->resolution_.y);
	//
	//	RectangleHelio *recthelio = dynamic_cast<RectangleHelio *>(solar_scene.heliostats[i]);
	//
	//	int avg_index = 0;
	//	for (int i_gral = 0; i_gral < sizeof(helio_granularity) / sizeof(float); ++i_gral)
	//	{
	//		recthelio->pixel_length_ = helio_granularity[i_gral];
	//		for (int j = start_n; j < end_n; ++j)
	//		{
	//			for (int i_n_per_group = 0; i_n_per_group < sizeof(sun_shape_per_group) / sizeof(int); ++i_n_per_group)
	//			{
	//				solar_scene.sunray_->num_sunshape_lights_per_group_ = sun_shape_per_group[i_n_per_group];
	//				solar_scene.sunray_->CClear();
	//				for (int i_csr = 0; i_csr < sizeof(csrs) / sizeof(float); ++i_csr)
	//				{
	//					solar_scene.sunray_->csr_ = csrs[i_csr];
	//					for (int i_dist = 0; i_dist < sizeof(disturb_stds) / sizeof(float); ++i_dist)
	//					{
	//						solarenergy::disturb_std = disturb_stds[i_dist];
	//						// reset sunray
	//						SceneProcessor::set_sunray_content(*solar_scene.sunray_);
	//
	//						// clean screen to all 0s
	//						recv->Cclean_image_content();
	//
	//						// ray-tracing
	//						recthelio_ray_tracing(*solar_scene.sunray_,
	//							*recv,
	//							*recthelio,
	//							*solar_scene.grid0s[i],
	//							solar_scene.heliostats);
	//
	//						// Save result
	//						global_func::gpu2cpu(h_image, recv->d_image_, recv->resolution_.x*recv->resolution_.y);
	//						// Id, Ssub, rou, Nc
	//						float Id = solar_scene.sunray_->dni_;
	//						float Ssub = recthelio->pixel_length_*recthelio->pixel_length_;
	//						float rou = solarenergy::reflected_rate;
	//						int Nc = solar_scene.sunray_->num_sunshape_lights_per_group_;
	//						float Srec = recv->pixel_length_*recv->pixel_length_;
	//						float max = -1.0f, max_avg = -1.0f;
	//						for (int p = 0; p < recv->resolution_.x*recv->resolution_.y; ++p)
	//						{
	//							h_image[p] = h_image[p] * Id * Ssub * rou / Nc / Srec;
	//							h_avg_img[p] = (h_avg_img[p] * float(j) + h_image[p]) / float(j + 1);
	//
	//							if (max < h_image[p])
	//								max = h_image[p];
	//
	//							if (max_avg < h_avg_img[p])
	//								max_avg = h_avg_img[p];
	//						}
	//
	//						if (j < save_number)
	//						{
	//							string tmp = save_path;
	//							tmp.insert(tmp.size() - 9, to_string(i));
	//							tmp.insert(tmp.size() - 8, to_string(j));
	//							tmp.insert(tmp.size() - 7, to_string(helio_granularity[i_gral]));
	//							tmp.insert(tmp.size() - 6, to_string(sun_shape_per_group[i_n_per_group]));
	//							tmp.insert(tmp.size() - 5, to_string(csrs[i_csr]));
	//							tmp.insert(tmp.size() - 4, to_string(disturb_stds[i_dist]));
	//							ImageSaver::savetxt(tmp, recv->resolution_.x, recv->resolution_.y, h_image);
	//						}
	//						
	//						printf("No.%d\n", ++index);
	//					}
	//				}
	//			}
	//			printf("(%d,\t%d)\n", i, j);
	//		}
	//	}
	//}
	//
	//delete[] h_image;
	//h_image = nullptr;
	//
	//delete[] h_avg_img;
	//h_avg_img = nullptr;



	RectangleHelio *recthelio = dynamic_cast<RectangleHelio *>(solar_scene.heliostats[24]);
	recthelio_ray_tracing(*solar_scene.sunray_,
							*solar_scene.receivers[0],
							*recthelio,
							*solar_scene.grid0s[0],
							solar_scene.heliostats);

	float *h_image = nullptr;
	global_func::gpu2cpu(h_image, solar_scene.receivers[0]->d_image_, solar_scene.receivers[0]->resolution_.x*solar_scene.receivers[0]->resolution_.y);
	// Id, Ssub, rou, Nc
	float Id=solar_scene.sunray_->dni_;
	float Ssub = recthelio->pixel_length_*recthelio->pixel_length_;
	float rou = solarenergy::reflected_rate;
	int Nc = solar_scene.sunray_->num_sunshape_lights_per_group_;
	float Srec = solar_scene.receivers[0]->pixel_length_*solar_scene.receivers[0]->pixel_length_;
	float max = -1.0f;
	for (int i = 0; i < solar_scene.receivers[0]->resolution_.x*solar_scene.receivers[0]->resolution_.y; ++i)
	{
		h_image[i] = h_image[i] * Id * Ssub * rou / Nc/ Srec;
	
		if (max < h_image[i])
			max = h_image[i];
	}
	
	// Save image	
	ImageSaver::savetxt("../result/24th-128.txt", solar_scene.receivers[0]->resolution_.x, solar_scene.receivers[0]->resolution_.y, h_image);
}