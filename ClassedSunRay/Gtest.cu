#include "hip/hip_runtime.h"
#include "Gtest.cuh"
#include "scene_instance_process.h"
#include "recthelio_tracing.h"
#include "image_save.h"
#include "image_smooth.cuh"

#include <sstream>
#include <chrono>


namespace tmp
{
	template<typename T>
	void init_matrix(T *matrix, int size)
	{
		for (int i = 0; i < size; ++i)
			matrix[i] = 0;
	}

	void save_array(string filename, float *array, int size)
	{
		ofstream out(filename.c_str());
		for (int i = 0; i < size; ++i)
			out << array[i] << endl;
		out.close();
	}
};

void test(SolarScene &solar_scene)
{
	////// helios
	////float helio_granularity[] = { 0.01f, 0.02f };
	//
	////// rays
	////int sun_shape_per_group[] = { 1024, 2048 };
	////float csrs[] = { 0.1f, 0.15f, 0.2f };	
	////float disturb_stds[] = { 0.001,0.002,0.003 };
	//
	//// helios
	//float helio_granularity[] = { 0.01f };
	////float helio_granularity[] = { 0.02f, 0.05f, 0.1f };
	//
	//// rays
	//int sun_shape_per_group[] = { 128 };
	////int sun_shape_per_group[] = {512};
	//float csrs[] = { 0.1f };
	////float csrs[] = { 0.2f, 0.3f };
	//float disturb_stds[] = { 0.001 };
	////float disturb_stds[] = { 0.003 };
	//
	//int start_n = 0, end_n = 50, save_number = 2000;
	//string save_path("../result//try//128-128-_____.txt");
	//
	//float *h_image = nullptr;
	//
	//int index = 0;
	//Receiver *recv = dynamic_cast<RectangleReceiver *>(solar_scene.receivers[0]);
	//
	//
	//float *h_avg_img = new float[recv->resolution_.x*recv->resolution_.y];
	//tmp::init_matrix(h_avg_img, recv->resolution_.x*recv->resolution_.y);
	//
	////int helio_num[] = { 4, 14, 24, 34,
	////					20, 22, 26, 28, 29 };
	//int helio_num[] = { 24 };
	//
	//for (int id = 0; id < sizeof(helio_num) / sizeof(helio_num[0]); ++id)
	//{
	//	int i = helio_num[id];
	//	tmp::init_matrix(h_avg_img, recv->resolution_.x*recv->resolution_.y);
	//
	//	RectangleHelio *recthelio = dynamic_cast<RectangleHelio *>(solar_scene.heliostats[i]);
	//
	//	int avg_index = 0;
	//	for (int i_gral = 0; i_gral < sizeof(helio_granularity) / sizeof(float); ++i_gral)
	//	{
	//		recthelio->pixel_length_ = helio_granularity[i_gral];
	//		for (int j = start_n; j < end_n; ++j)
	//		{
	//			for (int i_n_per_group = 0; i_n_per_group < sizeof(sun_shape_per_group) / sizeof(int); ++i_n_per_group)
	//			{
	//				solar_scene.sunray_->num_sunshape_lights_per_group_ = sun_shape_per_group[i_n_per_group];
	//				solar_scene.sunray_->CClear();
	//				for (int i_csr = 0; i_csr < sizeof(csrs) / sizeof(float); ++i_csr)
	//				{
	//					solar_scene.sunray_->csr_ = csrs[i_csr];
	//					for (int i_dist = 0; i_dist < sizeof(disturb_stds) / sizeof(float); ++i_dist)
	//					{
	//						solarenergy::disturb_std = disturb_stds[i_dist];
	//						// reset sunray
	//						SceneProcessor::set_sunray_content(*solar_scene.sunray_);
	//
	//						// clean screen to all 0s
	//						recv->Cclean_image_content();
	//
	//						// ray-tracing
	//						recthelio_ray_tracing(*solar_scene.sunray_,
	//							*recv,
	//							*recthelio,
	//							*solar_scene.grid0s[i],
	//							solar_scene.heliostats);
	//
	//						// Save result
	//						global_func::gpu2cpu(h_image, recv->d_image_, recv->resolution_.x*recv->resolution_.y);
	//						// Id, Ssub, rou, Nc
	//						float Id = solar_scene.sunray_->dni_;
	//						float Ssub = recthelio->pixel_length_*recthelio->pixel_length_;
	//						float rou = solarenergy::reflected_rate;
	//						int Nc = solar_scene.sunray_->num_sunshape_lights_per_group_;
	//						float Srec = recv->pixel_length_*recv->pixel_length_;
	//						float max = -1.0f, max_avg = -1.0f;
	//						for (int p = 0; p < recv->resolution_.x*recv->resolution_.y; ++p)
	//						{
	//							h_image[p] = h_image[p] * Id * Ssub * rou / Nc / Srec;
	//							h_avg_img[p] = (h_avg_img[p] * float(j) + h_image[p]) / float(j + 1);
	//
	//							if (max < h_image[p])
	//								max = h_image[p];
	//
	//							if (max_avg < h_avg_img[p])
	//								max_avg = h_avg_img[p];
	//						}
	//
	//						if (j < save_number)
	//						{
	//							string tmp = save_path;
	//							tmp.insert(tmp.size() - 9, to_string(i));
	//							tmp.insert(tmp.size() - 8, to_string(j));
	//							tmp.insert(tmp.size() - 7, to_string(helio_granularity[i_gral]));
	//							tmp.insert(tmp.size() - 6, to_string(sun_shape_per_group[i_n_per_group]));
	//							tmp.insert(tmp.size() - 5, to_string(csrs[i_csr]));
	//							tmp.insert(tmp.size() - 4, to_string(disturb_stds[i_dist]));
	//							ImageSaver::savetxt(tmp, recv->resolution_.x, recv->resolution_.y, h_image);
	//						}
	//						
	//						printf("No.%d\n", ++index);
	//					}
	//				}
	//			}
	//			printf("(%d,\t%d)\n", i, j);
	//		}
	//	}
	//}
	//
	//delete[] h_image;
	//h_image = nullptr;
	//
	//delete[] h_avg_img;
	//h_avg_img = nullptr;

	string save_path("../result/24///24_.txt"); // e.g. - ../result/24/256/24-0.txt
	int helio_id[] = { 24 };
	int num_groups[] = { 32 };
	int run_times_start = 900, run_times_end = 901;

	// Smooth result
	int kernel_radius = 5;
	float trimmed_ratio = 0.02;
	float *h_image = nullptr;

	RectangleHelio *recthelio = dynamic_cast<RectangleHelio *>(solar_scene.heliostats[24]);
	Receiver *recv = dynamic_cast<RectangleReceiver *>(solar_scene.receivers[0]);

	// time
	auto start = std::chrono::high_resolution_clock::now();			 // nano-seconds
	auto elapsed = std::chrono::high_resolution_clock::now() - start;// nano-seconds
	long long total_time = 0, ray_gen_time = 0;
	long long time_tracing = 0, time_subcenter = 0, time_group_gen = 0;
	long long time_smooth = 0;

	recthelio->type = SubCenterType::Poisson;
	for (int i = 0; i < sizeof(num_groups) / sizeof(num_groups[0]); ++i)
	{
		solar_scene.sunray_->num_sunshape_groups_ = num_groups[i];
		for (int j = run_times_start; j < run_times_end; ++j)
		{
			// Reset the content of sun
			solar_scene.sunray_->CClear();
			start = std::chrono::high_resolution_clock::now();
			SceneProcessor::set_sunray_content(*solar_scene.sunray_);
			elapsed = std::chrono::high_resolution_clock::now() - start;
			ray_gen_time+= std::chrono::duration_cast<std::chrono::microseconds>(elapsed).count();

			// Clear result of receiver
			recv->Cclean_image_content();

			// Ray-tracing
			int num_subcenters = recthelio_ray_tracing(*solar_scene.sunray_,
				*recv, *recthelio,
				*solar_scene.grid0s[0],
				solar_scene.heliostats,
				time_tracing,
				time_subcenter,
				time_group_gen);
			elapsed = std::chrono::high_resolution_clock::now() - start;
			total_time += std::chrono::duration_cast<std::chrono::microseconds>(elapsed).count();

			//global_func::gpu2cpu(h_image, solar_scene.receivers[0]->d_image_, recv->resolution_.x*recv->resolution_.y);
			
			//// Non Smooth
			//float Id = solar_scene.sunray_->dni_;
			//float Ssub = recthelio->pixel_length_*recthelio->pixel_length_;
			//float rou = solarenergy::reflected_rate;
			//int Nc = solar_scene.sunray_->num_sunshape_lights_per_group_;
			//float Srec = recv->pixel_length_*recv->pixel_length_;
			//for (int k = 0; k < recv->resolution_.x * recv->resolution_.y; ++k)
			//	h_image[k] *= Id * Ssub * rou / Nc / Srec;
			//// Save image	
			//string tmp_path = save_path;
			//tmp_path.insert(tmp_path.size() - 9, to_string(num_groups[i]));
			//tmp_path.insert(tmp_path.size() - 8, "non_smooth");
			//tmp_path.insert(tmp_path.size() - 4, to_string(j));
			//ImageSaver::savetxt(tmp_path, recv->resolution_.x, recv->resolution_.y, h_image);

			// Smooth
			start = std::chrono::high_resolution_clock::now();
			ImageSmoother::image_smooth(recv->d_image_,
				kernel_radius, trimmed_ratio,
				recv->resolution_.x, recv->resolution_.y);
			elapsed = std::chrono::high_resolution_clock::now() - start;
			time_smooth += std::chrono::duration_cast<std::chrono::microseconds>(elapsed).count();
			total_time += std::chrono::duration_cast<std::chrono::microseconds>(elapsed).count();

			//global_func::gpu2cpu(h_image, solar_scene.receivers[0]->d_image_, recv->resolution_.x*recv->resolution_.y);
			//for (int k = 0; k < recv->resolution_.x*recv->resolution_.y; ++k)
			//	h_image[k] *= Id * Ssub * rou / Nc / Srec;
			//tmp_path = save_path;//"../result/24////24-.txt"
			//tmp_path.insert(tmp_path.size() - 9, to_string(num_groups[i]));
			//tmp_path.insert(tmp_path.size() - 8, "smooth");
			//tmp_path.insert(tmp_path.size() - 4, to_string(j));
			//ImageSaver::savetxt(tmp_path, recv->resolution_.x, recv->resolution_.y, h_image);
			//cout << tmp_path << endl;
		}
		std::cout << to_string(num_groups[i]) << endl;
		std::cout << "Total Average Time:\t" + to_string(double(total_time / (run_times_end - run_times_start))) << endl;
		std::cout << "Rays Generation Time:\t" + to_string(double(ray_gen_time / (run_times_end - run_times_start))) << endl;
		std::cout << "Subcenter Generation Time:\t" + to_string(double(time_subcenter / (run_times_end - run_times_start))) << endl;
		std::cout << "Groups Generation Time:\t" + to_string(double(time_group_gen / (run_times_end - run_times_start))) << endl;
		std::cout << "Tracing Time:\t" + to_string(double(time_tracing / (run_times_end - run_times_start))) << endl;
		std::cout << "Smooth Time:\t" + to_string(double(time_smooth / (run_times_end - run_times_start))) << endl;
		std::cout << endl;
	}


	delete[] h_image;
	h_image = nullptr;
	solar_scene.sunray_->CClear();
	recv->Cclean_image_content();
	
	//RectangleHelio *recthelio = dynamic_cast<RectangleHelio *>(solar_scene.heliostats[24]);
	//int num_subcenters =recthelio_ray_tracing(*solar_scene.sunray_,
	//						*solar_scene.receivers[0],
	//						*recthelio,
	//						*solar_scene.grid0s[0],
	//						solar_scene.heliostats);
	//// Smooth result
	//int kernel_radius = 5;
	//float trimmed_ratio = 0.03;
	//ImageSmoother::image_smooth(solar_scene.receivers[0]->d_image_,
	//	kernel_radius, trimmed_ratio,
	//	solar_scene.receivers[0]->resolution_.x, solar_scene.receivers[0]->resolution_.y);

	//float *h_image = nullptr;
	//global_func::gpu2cpu(h_image, solar_scene.receivers[0]->d_image_, solar_scene.receivers[0]->resolution_.x*solar_scene.receivers[0]->resolution_.y);
	//// Id, Ssub, rou, Nc
	//float Id=solar_scene.sunray_->dni_;
	////float Ssub = recthelio->pixel_length_*recthelio->pixel_length_;
	//float rou = solarenergy::reflected_rate;
	//int Nc = solar_scene.sunray_->num_sunshape_lights_per_group_;
	////float Srec = solar_scene.receivers[0]->pixel_length_*solar_scene.receivers[0]->pixel_length_;
	//float max = -1.0f;

	//int num_recv_m2 = (1 / solar_scene.receivers[0]->pixel_length_)*(1 / solar_scene.receivers[0]->pixel_length_);
	//float w = recthelio->size_.x - recthelio->gap_.x*(recthelio->row_col_.y - 1);
	//float h = recthelio->size_.z - recthelio->gap_.y*(recthelio->row_col_.x - 1);
	//float multiplier = (w*h*float(num_recv_m2)*Id * rou) / float(Nc*num_subcenters);

	//for (int i = 0; i < solar_scene.receivers[0]->resolution_.x*solar_scene.receivers[0]->resolution_.y; ++i)
	//{
	//	//h_image[i] = h_image[i] * Id * Ssub * rou / Nc/ Srec;

	//	// Ssub / Srec = Shelio_area* (1/Srec) / num_subcenters
	//	h_image[i] *= multiplier;	
	//	if (max < h_image[i])
	//		max = h_image[i];
	//}
	//
	//// Save image	
	//ImageSaver::savetxt("../result/24th-64group-1024pergroup-poisson-smoothed.txt", solar_scene.receivers[0]->resolution_.x, solar_scene.receivers[0]->resolution_.y, h_image);
}