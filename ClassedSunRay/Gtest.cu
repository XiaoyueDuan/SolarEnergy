#include "hip/hip_runtime.h"
#include "Gtest.cuh"
 
void test(SolarScene &solar_scene)
{
	// Receiver
	//solar_scene.receivers[0]->normal_ = make_float3(1.0f, 0.0f, 0.0f);
	solar_scene.receivers[0]->CInit(100);
	
	float3 dir = make_float3(0.0f, 0.0f, - 1.0f);
	float3 orig = make_float3(4.0f, 104.0f, 100.0f);

	float t, u, v;
	solar_scene.receivers[0]->GIntersect(orig, dir, t, u, v);
	
	// Heliostat
	RectangleHelio *rectangle_helio =dynamic_cast<RectangleHelio *>(solar_scene.heliostats[0]);
	rectangle_helio->Cset_sub_row_col(0.01f);
	rectangle_helio->init(solar_scene.receivers[0]->pos_);

	// Grid

}