#include "hip/hip_runtime.h"
#include "Gtest.cuh"
#include "scene_instance_process.h"
#include "recthelio_tracing.h"
#include "image_save.h"

#include <sstream>

namespace tmp {
	// Step 2: Generate micro-heliostats' normals
	__global__ void map_microhelio_normals(float3 *d_microhelio_normals, const float3 *d_microhelio_centers,
		float3 normal,
		const size_t size)
	{
		unsigned long long int myId = global_func::getThreadId();
		if (myId >= size)
			return;

		d_microhelio_normals[myId] = normal;
	}
	
	//// Step 2: Generate micro-heliostats' normals
	//__global__ void map_microhelio_normals(float3 *d_microhelio_normals, const float3 *d_microhelio_centers,
	//	float3 normal,
	//	const size_t size)
	//{
	//	unsigned long long int myId = global_func::getThreadId();
	//	if (myId >= size)
	//		return;

	//	float3 local_normal = d_microhelio_normals[myId];
	//	local_normal = global_func::local2world(local_normal, normal);
	//	local_normal = normalize(local_normal);
	//	d_microhelio_normals[myId] = local_normal;
	//}

	// Step 3: Transform local micro-helio center to world postion
	__global__ void map_microhelio_center2world(float3 *d_microhelio_world_centers, float3 *d_microhelio_local_centers,
		const float3 normal, const float3 world_pos,
		const size_t size)
	{
		unsigned long long int myId = global_func::getThreadId();
		if (myId >= size)
			return;

		float3 local = d_microhelio_local_centers[myId];
		local = global_func::local2world(local, normal);		// Then Rotate
		local = global_func::transform(local, world_pos);		// Translation to the world system
		d_microhelio_world_centers[myId] = local;
	}
}


void test(SolarScene &solar_scene)
{
	solar_scene.InitContent();
	
	RectangleHelio *recthelio = dynamic_cast<RectangleHelio *>(solar_scene.heliostats[0]);
	recthelio_ray_tracing(*solar_scene.sunray_,
							*solar_scene.receivers[0],
							*recthelio,
							*solar_scene.grid0s[0],
							solar_scene.heliostats);
	
	float *h_image = nullptr;
	global_func::gpu2cpu(h_image, solar_scene.receivers[0]->d_image_, solar_scene.receivers[0]->resolution_.x*solar_scene.receivers[0]->resolution_.y);
	// Id, Ssub, rou, Nc
	float Id=solar_scene.sunray_->dni_;
	float Ssub = recthelio->pixel_length_*recthelio->pixel_length_;
	float rou = solarenergy::reflected_rate;
	int Nc = solar_scene.sunray_->num_sunshape_lights_per_group_;
	float Srec = solar_scene.receivers[0]->pixel_length_*solar_scene.receivers[0]->pixel_length_;
	float max = -1.0f;
	for (int i = 0; i < solar_scene.receivers[0]->resolution_.x*solar_scene.receivers[0]->resolution_.y; ++i)
	{
		h_image[i] = h_image[i] * Id * Ssub * rou / Nc/ Srec;
	
		if (max < h_image[i])
			max = h_image[i];
	}
	
	// Save image	
	ImageSaver::savetxt("../result/validate-with-paper.txt", solar_scene.receivers[0]->resolution_.x, solar_scene.receivers[0]->resolution_.y, h_image);

	//// load centers & normals
	//ifstream in("../userData/check_with_real_data/sandun.txt");
	//if (in.fail())
	//{
	//	cout << "The file does not exist!" << endl;
	//	return;
	//}

	//// step 2: prepare for data loading	
	//streambuf *cinbuf = cin.rdbuf();	//save old buf
	//cin.rdbuf(in.rdbuf());				//redirect cin to file
	//istringstream ss;
	//
	//int N;
	//cin >> N;	
	//float3 *h_microhelio_normals = new float3[N];		// micro-heliostat's normal
	//float3 *h_microhelio_origs = new float3[N];			// micro-heliostat's origins
	//
	//string line;
	//for (int i = 0; i < N; )
	//{
	//	getline(cin, line);
	//	if (line.empty() || line.substr(0, 2) == "//")	// empty or comment
	//		continue;

	//	ss.str(line); ss.clear();
	//	ss >> h_microhelio_origs[i].x;
	//	ss >> h_microhelio_origs[i].y;
	//	ss >> h_microhelio_origs[i].z;
	//	ss >> h_microhelio_normals[i].x;
	//	ss >> h_microhelio_normals[i].y;
	//	ss >> h_microhelio_normals[i].z;
	//	++i;
	//}
	//cin.rdbuf(cinbuf);

	//// transfer to gpu
	//float3 *d_microhelio_normals = nullptr;		// micro-heliostat's normal
	//float3 *d_microhelio_origs = nullptr;			// micro-heliostat's origins
	//global_func::cpu2gpu(d_microhelio_normals, h_microhelio_normals, N);
	//global_func::cpu2gpu(d_microhelio_origs, h_microhelio_origs, N);

	//delete[] h_microhelio_normals;
	//delete[] h_microhelio_origs;
	//h_microhelio_normals = nullptr;
	//h_microhelio_origs = nullptr;
	//
	//// tracing 1st
	//RectGrid *rectgrid0 = dynamic_cast<RectGrid *> (solar_scene.grid0s[0]);
	//RectangleReceiver *rect_receiver0 = dynamic_cast<RectangleReceiver *> (solar_scene.receivers[0]);

	//// d_helio_vertexs
	//int start_pos = rectgrid0->start_helio_pos_;
	//int end_pos = start_pos + rectgrid0->num_helios_;
	//float3 *d_helio_vertexs = nullptr;
	//set_helios_vertexes(solar_scene.heliostats, start_pos, end_pos, d_helio_vertexs);
	//hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

	//// d_microhelio_groups
	//int *d_microhelio_groups = nullptr;
	//set_microhelio_groups(d_microhelio_groups, solar_scene.sunray_->num_sunshape_groups_, N);
	//hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

	//
	//// micro-center local2world
	//int nThreads;
	//dim3 nBlocks;
	//global_func::setThreadsBlocks(nBlocks, nThreads, N);
	//tmp::map_microhelio_center2world <<<nBlocks, nThreads >>> (d_microhelio_origs, d_microhelio_origs,
	//	solar_scene.heliostats[0]->normal_, solar_scene.heliostats[0]->pos_, N);

	//// micro-normal local2world
	//tmp::map_microhelio_normals << <nBlocks, nThreads >> > 
	//	(d_microhelio_normals, d_microhelio_origs, solar_scene.heliostats[0]->normal_, N);

	//// Tracing	
	//nThreads = 256;
	//nBlocks;
	//global_func::setThreadsBlocks(nBlocks, nThreads, N*(solar_scene.sunray_->num_sunshape_lights_per_group_), true);
	//map_tracing <<<nBlocks, nThreads >>>(*solar_scene.sunray_, *rectgrid0, *rect_receiver0,
	//	d_helio_vertexs, d_microhelio_normals, d_microhelio_origs, d_microhelio_groups,
	//	N);
	//hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());


	//float *h_image = nullptr;
	//global_func::gpu2cpu(h_image, solar_scene.receivers[0]->d_image_, solar_scene.receivers[0]->resolution_.x*solar_scene.receivers[0]->resolution_.y);
	//// Id, Ssub, rou, Nc
	//float Id=solar_scene.sunray_->dni_;
	//float Ssub = 0.005;
	//float rou = solarenergy::reflected_rate;
	//int Nc = solar_scene.sunray_->num_sunshape_lights_per_group_;
	//float Srec = solar_scene.receivers[0]->pixel_length_*solar_scene.receivers[0]->pixel_length_;
	//float max = -1.0f;
	//for (int i = 0; i < solar_scene.receivers[0]->resolution_.x*solar_scene.receivers[0]->resolution_.y; ++i)
	//{
	//	h_image[i] = h_image[i] * Id * Ssub * rou / Nc/ Srec;
	//
	//	if (max < h_image[i])
	//		max = h_image[i];
	//}
	//
	//// Save image	
	//ImageSaver::savetxt("real_1.txt", solar_scene.receivers[0]->resolution_.x, solar_scene.receivers[0]->resolution_.y, h_image);
	//delete[] h_image;
	//h_image = nullptr;

	////// tracing 1st
	////RectGrid *rectgrid0 = dynamic_cast<RectGrid *> (solar_scene.grid0s[1]);
	////RectangleReceiver *rect_receiver0 = dynamic_cast<RectangleReceiver *> (solar_scene.receivers[1]);

	////// d_helio_vertexs
	////int start_pos = rectgrid0->start_helio_pos_;
	////int end_pos = start_pos + rectgrid0->num_helios_;
	////float3 *d_helio_vertexs = nullptr;
	////set_helios_vertexes(solar_scene.heliostats, start_pos, end_pos, d_helio_vertexs);
	////hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

	////// d_microhelio_groups
	////int *d_microhelio_groups = nullptr;
	////set_microhelio_groups(d_microhelio_groups, solar_scene.sunray_->num_sunshape_groups_, N);
	////hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());


	////// micro-center local2world
	////int nThreads;
	////dim3 nBlocks;
	////global_func::setThreadsBlocks(nBlocks, nThreads, N);
	////tmp::map_microhelio_center2world << <nBlocks, nThreads >> > (d_microhelio_origs, d_microhelio_origs,
	////	solar_scene.heliostats[1]->normal_, solar_scene.heliostats[1]->pos_, N);

	////// micro-normal local2world
	////tmp::map_microhelio_normals << <nBlocks, nThreads >> >
	////	(d_microhelio_normals, d_microhelio_origs, solar_scene.heliostats[1]->normal_, N);

	////// Tracing	
	////nThreads = 256;
	////nBlocks;
	////global_func::setThreadsBlocks(nBlocks, nThreads, N*(solar_scene.sunray_->num_sunshape_lights_per_group_), true);
	////map_tracing << <nBlocks, nThreads >> >(*solar_scene.sunray_, *rectgrid0, *rect_receiver0,
	////	d_helio_vertexs, d_microhelio_normals, d_microhelio_origs, d_microhelio_groups,
	////	N);
	////hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());


	////float *h_image = nullptr;
	////global_func::gpu2cpu(h_image, solar_scene.receivers[1]->d_image_, solar_scene.receivers[1]->resolution_.x*solar_scene.receivers[1]->resolution_.y);
	////// Id, Ssub, rou, Nc
	////float Id = solar_scene.sunray_->dni_;
	////float Ssub = 0.005;
	////float rou = solarenergy::reflected_rate;
	////int Nc = solar_scene.sunray_->num_sunshape_lights_per_group_;
	////float Srec = solar_scene.receivers[1]->pixel_length_*solar_scene.receivers[1]->pixel_length_;
	////float max = -1.0f;
	////for (int i = 0; i < solar_scene.receivers[1]->resolution_.x*solar_scene.receivers[1]->resolution_.y; ++i)
	////{
	////	h_image[i] = h_image[i] * Id * Ssub * rou / Nc / Srec;

	////	if (max < h_image[i])
	////		max = h_image[i];
	////}

	////// Save image	
	////ImageSaver::savetxt("real_2.txt", solar_scene.receivers[1]->resolution_.x, solar_scene.receivers[1]->resolution_.y, h_image);
	////delete[] h_image;
	////h_image = nullptr;
}